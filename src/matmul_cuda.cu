#include "hip/hip_runtime.h"
#include <filesystem>
#include <vector>
#include <random>
#include <cstdlib>

#include "./utils/chrono.cpp"
#include "hip/hip_runtime.h"

static const std::string file = std::filesystem::path(__FILE__).filename();

struct Matrix {
    float* data;
    std::size_t size;
};

__global__ void matmulKer(float* a, float* b, float* c, std::size_t n)
{
    std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    std::size_t j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= n || j >= n)
        return;

    for (std::size_t k = 0; k < n; k++)
        c[i * n + j] += a[i * n + k] * b[k * n + j];
}

Matrix random_matrix(std::size_t n)
{
    auto mat = Matrix{
        .data = new float[n * n] {0.0},
        .size = n
    };

    std::mt19937 gen(std::random_device{}());
    std::uniform_real_distribution<float> urd(0.0, 1.0);
    for (std::size_t i = 0; i < n * n; i++)
        mat.data[i] = urd(gen);
    
    float* dMat; std::size_t size = sizeof(float) * n * n;
    hipMalloc(&dMat, size);
    hipMemcpy(dMat, mat.data, size, hipMemcpyHostToDevice);
    delete mat.data; mat.data = dMat;

    return mat;
}

Matrix matmul(const Matrix& a, const Matrix& b)
{
    auto n = a.size;

    auto c = Matrix{
        .data = new float[n * n] {0.0},
        .size = n
    };

    float* dC; std::size_t size = sizeof(float) * n * n;
    hipMalloc(&dC, size);
    hipMemcpy(dC, c.data, size, hipMemcpyHostToDevice);
    delete c.data; c.data = dC;

    unsigned int dimBlock = 32;
    unsigned int dimGrid = n / dimBlock;
    if (dimGrid * dimBlock < n) dimGrid += 1;

    dim3 numBlocks { dimGrid, dimGrid };
    dim3 numThreads { dimBlock, dimBlock };
    matmulKer<<<numBlocks, numThreads>>>(a.data, b.data, c.data, n);
    hipDeviceSynchronize();
    
    return c;
}

int main()
{
    const std::size_t MATRIX_SIZE = std::strtoul(std::getenv("YOB_DEMO_MS"), nullptr, 10);

    auto a = random_matrix(MATRIX_SIZE);
    auto b = random_matrix(MATRIX_SIZE);
    Matrix c {};

    auto chr = Chrono();
    {
        auto _ = chr.start();
        c = matmul(a, b);
    }
    chr.print(file);

    hipFree(a.data);
    hipFree(b.data);
    hipFree(c.data);
    return 0;
}